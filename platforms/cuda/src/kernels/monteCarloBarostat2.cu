#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is the cuda version of atom position rescaling code for the           *
 * ansiotropical barostat as part of the OpenMM plugins.                      *
 *                                                                            *
 * Portions copyright (c) 2013 the University of Michigan and the Authors.    *
 * Authors: Shuai Wei                                                         *
 * Contributors: Charles L. Brooks III and Michael Garrahan                   *
 *                                                                            *
 * Permission is hereby granted, free of charge, to any person obtaining a    *
 * copy of this software and associated documentation files (the "Software"), *
 * to deal in the Software without restriction, including without limitation  *
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,   *
 * and/or sell copies of the Software, and to permit persons to whom the      *
 * Software is furnished to do so, subject to the following conditions:       *
 *                                                                            *
 * The above copyright notice and this permission notice shall be included in *
 * all copies or substantial portions of the Software.                        *
 *                                                                            *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR *
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,   *
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL    *
 * THE AUTHORS, CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,    *
 * DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR      *
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE  *
 * USE OR OTHER DEALINGS IN THE SOFTWARE.                                     *
 * -------------------------------------------------------------------------- */

extern "C" __global__ void scalePositions2(double4 scale, int numMolecules, real4 periodicBoxSize, real4 invPeriodicBoxSize, real4* __restrict__ posq,
        const int* __restrict__ moleculeAtoms, const int* __restrict__ moleculeStartIndex) {
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < numMolecules; index += blockDim.x*gridDim.x) {
        int first = moleculeStartIndex[index];
        int last = moleculeStartIndex[index+1];
        int numAtoms = last-first;

        // Find the center of each molecule.

        real3 center = make_real3(0, 0, 0);
        for (int atom = first; atom < last; atom++) {
            real4 pos = posq[moleculeAtoms[atom]];
            center.x += pos.x;
            center.y += pos.y;
            center.z += pos.z;
        }
        real invNumAtoms = RECIP(numAtoms);
        center.x *= invNumAtoms;
        center.y *= invNumAtoms;
        center.z *= invNumAtoms;

        // Move it into the first periodic box.
        int xcell;
        int ycell;
        int zcell;
        xcell = (int) floor(center.x*invPeriodicBoxSize.x);
        ycell = (int) floor(center.y*invPeriodicBoxSize.y);
        zcell = (int) floor(center.z*invPeriodicBoxSize.z);

        real3 delta = make_real3(xcell*periodicBoxSize.x, ycell*periodicBoxSize.y, zcell*periodicBoxSize.z);

        center.x -= delta.x;
        center.y -= delta.y;
        center.z -= delta.z;

        // Now scale the position of the molecule center.
        
        delta.x = center.x*(scale.x-1)-delta.x;
        delta.y = center.y*(scale.y-1)-delta.y;
        delta.z = center.z*(scale.z-1)-delta.z;
        for (int atom = first; atom < last; atom++) {
            real4 pos = posq[moleculeAtoms[atom]];
            pos.x += delta.x;
            pos.y += delta.y;
            pos.z += delta.z;
            posq[moleculeAtoms[atom]] = pos;
        }
    }
}

